
#include "hip/hip_runtime.h"
#include <iostream>

//如果使用vs 新建项目 注意有多个main函数情况
int main()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; i++)
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
		std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
		std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
		std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
		std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
		std::cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
		std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
		std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
		std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;
		std::cout << "======================================================" << std::endl;

	}
	return 0;
}

/*
使用GPU device 0: GeForce RTX 2070
设备全局内存总量： 8192MB
SM的数量：36
每个线程块的共享内存大小：48 KB
每个线程块的最大线程数：1024
设备上一个线程块（Block）种可用的32位寄存器数量： 65536
每个EM的最大线程数：1024
每个EM的最大线程束数：32
设备上多处理器的数量： 36
*/